float *tmp01,*tmp02,*tmp03,*tmp04,*tmp05;

hipMalloc((void **)&tmp02,SIZE);
hipMemcpy(tmp02,tmp01,SIZE,hipMemcpyHostToDevice);

foo <<<N_BLOCKS,BLOCK_SIZE>>> (tmp02, N);
tmp03 = tmp02;

bar <<<N_BLOCKS,BLOCK_SIZE>>> (tmp03,N);
tmp04 = tmp03;

tmp05 = malloc(SIZE * sizeof(float));
hipMemcpy(tmp05,tmp04,SIZE,hipMemcpyHostToDevice);
