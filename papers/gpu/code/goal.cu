// Here is what we want to generate. From:
// http://llpanorama.wordpress.com/2008/05/21/my-first-cuda-program/

// Build with something like:
// nvcc -I/usr/local/cuda/include/ -L/usr/local/cuda/lib/ goal.cu

#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel that executes on the CUDA device
__global__ void square_array(float *a, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N) a[idx] = a[idx] * a[idx];
}

void handle_error(hipError_t err) {
  printf("Error: %s\n",hipGetErrorString(err));
  exit(1);
}

int main() {  
  // Pointer to host and device arrays
  float *a_h, *a_d;
  
  // Number of elements in arrays
  const int N = 10;
  size_t size = N * sizeof(float);
  
  hipError_t err;
  
  int deviceCount = 0;
  err = hipGetDeviceCount(&deviceCount);
  if(err) handle_error(err);
  printf("Found %d CUDA devices\n",deviceCount);
  
  // Allocate arrays
  a_h = (float *)malloc(size);
  err = hipMalloc((void **) &a_d, size);
  if(err) handle_error(err);
  
  // Initialize host array
  for(int i=0; i<N; i++) {
    a_h[i] = (float)i;
  }
  
  // Copy host to device
  err = hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  if(err) handle_error(err);
  
  // Do calculation on device
  int block_size = 4;
  int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
  square_array <<< n_blocks, block_size >>> (a_d, N);
  
  // Copy from device to host
  err = hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);
  if(err) handle_error(err);
  
  // Print results
  for (int i=0; i<N; i++) {
    printf("%d %f\n", i, a_h[i]);
  }
  
  // Cleanup
  free(a_h);
  hipFree(a_d);
}
