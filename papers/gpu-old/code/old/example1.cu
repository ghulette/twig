#include "hip/hip_runtime.h"
// Unoptimized version

#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel that executes on the CUDA device
__global__ void times_two(int *a, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N) a[idx] *= 2;
}

// Kernel that executes on the CUDA device
__global__ void plus_one(int *a, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx<N) a[idx] += 1;
}

void handle_error(hipError_t err) {
  printf("Error: %s\n",hipGetErrorString(err));
  exit(1);
}

int main() {
  // Result codes
  hipError_t err;
  
  // Pointer to host and device arrays
  int *a_h, *a_d;
  
  // Number of elements in arrays
  const int N = 128;
  size_t size = N * sizeof(int);
  int block_size = 4;
  int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
  
  // Query devices
  int deviceCount = 0;
  err = hipGetDeviceCount(&deviceCount);
  if(err) handle_error(err);
  printf("Found %d CUDA devices\n",deviceCount);
  
  hipDeviceProp_t props;
  for(int i=0; i<deviceCount; i++) {
    err = hipGetDeviceProperties(&props,0);
    if(err) handle_error(err);
    printf("Using: %s\n",props.name);
  }
  
  // Allocate and initialize host array
  a_h = (int *)malloc(size);
  for(int i=0; i<N; i++) {
    a_h[i] = i;
  }
  
  // Allocate CUDA array
  err = hipMalloc((void **) &a_d, size);
  if(err) handle_error(err);
  
// ===========================
// =        Block #1         =
// ===========================
  // Copy host to device
  err = hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  if(err) handle_error(err);
  // Do calculation on device
  times_two <<< n_blocks, block_size >>> (a_d, N);
  // Wait for device to finish
  err = hipDeviceSynchronize();
  if(err) handle_error(err);
  // Copy from device to host
  err = hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);
  if(err) handle_error(err);

// ===========================
// =        Block #2         =
// ===========================
  // Copy host to device
  err = hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  if(err) handle_error(err);
  // Do calculation on device
  plus_one <<< n_blocks, block_size >>> (a_d, N);
  // Wait for device to finish
  err = hipDeviceSynchronize();
  if(err) handle_error(err);
  // Copy from device to host
  err = hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);
  if(err) handle_error(err);

  // Print results
  for(int i=0; i<N; i++) {
    printf("%d %d\n", i, a_h[i]);
  }
  
  // Cleanup
  free(a_h);
  hipFree(a_d);
}
