#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

const int N = 10;
const size_t SIZE = N * sizeof(float);
const int BLOCK_SIZE = 4;
const int N_BLOCKS = N / BLOCK_SIZE + (N % BLOCK_SIZE == 0 ? 0:1);

__global__ void foo(float *a, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<N) {
    a[idx] = a[idx] * a[idx];
  }
}

__global__ void bar(float *a, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx<N) {
    a[idx] = a[idx] + 1;
  }
}

#include "twig.cu"

int main(void) {
  float *input, *result;
  input = (float *)malloc(SIZE);
  for(int i=0; i < N; i++) {
    input[i] = (float)i;
  }
  result = twig_gen_fun(input);
  for(int i=0; i < N; i++) {
    printf("%d -> %f\n", i, result[i]);
  }
  return 0;
}
