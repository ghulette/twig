float *twig_gen_fun(float *in) {
  float *tmp01,*tmp02,*tmp03,*tmp04,*tmp05;
  tmp01 = in;
  hipMalloc((void **)&tmp02,SIZE);
  hipMemcpy(tmp02,tmp01,SIZE,hipMemcpyHostToDevice);
  foo <<<N_BLOCKS,BLOCK_SIZE>>> (tmp02,N);
  tmp03 = tmp02;
  bar <<<N_BLOCKS,BLOCK_SIZE>>> (tmp03,N);
  tmp04 = tmp03;
  tmp05 = (float *)malloc(SIZE * sizeof(float));
  hipMemcpy(tmp05,tmp04,SIZE,hipMemcpyHostToDevice);
  return tmp05;
}

float *twig_gen_fun02(float *in) {
  float *tmp01,*tmp02,*tmp03,*tmp04,*tmp05,*tmp06,*tmp07;
  tmp01 = in;
  hipMalloc((void **)&tmp02,SIZE);
  hipMemcpy(tmp02,tmp01,SIZE,hipMemcpyHostToDevice);
  foo <<<N_BLOCKS,BLOCK_SIZE>>> (tmp02,N);
  tmp03 = tmp02;
  tmp04 = (float *)malloc(SIZE * sizeof(float));
  hipMemcpy(tmp04,tmp03,SIZE,hipMemcpyHostToDevice);
  hipMalloc((void **)&tmp05,SIZE);
  hipMemcpy(tmp05,tmp04,SIZE,hipMemcpyHostToDevice);
  bar <<<N_BLOCKS,BLOCK_SIZE>>> (tmp05,N);
  tmp06 = tmp05;
  tmp07 = (float *)malloc(SIZE * sizeof(float));
  hipMemcpy(tmp07,tmp06,SIZE,hipMemcpyHostToDevice);
  return tmp07;
}
